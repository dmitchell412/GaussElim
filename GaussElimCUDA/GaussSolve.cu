#include "hip/hip_runtime.h"

/*
 * Device code
 */
__global__ 
void GaussSolve(
         int const Nsize,
         double* d_Aug,
         double* d_Piv)
{
    // Assign matrix elements to blocks and threads
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

    // Parallel forward elimination
    for (int k = 0; k < Nsize-1; k++)
    {
        d_Piv[i] = d_Aug[i][k]/d_Aug[k][k];
        __syncthreads();
        if ((i>k) && (i<Nsize) && (j>=k) && (j<=Nsize))
            d_Aug[i][j] -= d_Piv[i]*d_Aug[k][j];
        __syncthreads();
    }
}
