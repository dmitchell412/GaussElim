
#include <hip/hip_runtime.h>

/*
 * Device code
 */
__global__ 
void GaussSolve(
         int const Nsize,
         double* d_Aug,
         double* d_Piv)
{
    // Assign matrix elements to blocks and threads
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

    // Parallel forward elimination
    for (int k = 0; k < Nsize-1; k++)
    {
        d_Piv[i] = d_Aug[Nsize*i+k]/d_Aug[Nsize*k+k];
        __syncthreads();
        if ((i>k) && (i<Nsize) && (j>=k) && (j<=Nsize))
            d_Aug[Nsize*i+j] -= d_Piv[i]*d_Aug[Nsize*k+j];
        __syncthreads();
    }
}
