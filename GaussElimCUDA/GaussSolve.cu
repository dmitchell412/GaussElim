
#include <hip/hip_runtime.h>

/*
 * Device code
 */
__global__ 
void GaussSolve(
         int const Nsize,
         const double* d_Matrix,
         const double* d_RHS,
               double* d_Soln)
{
    /* Calculate the global linear index, assuming a 1-d grid. */
    int const idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < Nsize)
      printf("idx=%d A[%d][%d]=%f b[%d]=%f\n",idx,idx,idx,d_Matrix[idx+Nsize*idx], idx,d_RHS[idx]);
}
