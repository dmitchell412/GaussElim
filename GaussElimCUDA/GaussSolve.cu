
#include <hip/hip_runtime.h>

/*
 * Device code
 */

__global__ 
void GaussSolve(
         int const Nsize,
         double* d_Aug,
         double* d_Piv)
{
    // Assign matrix elements to blocks and threads
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    // Parallel forward elimination
    for (int k = 0; k < Nsize-1; k++)
    {
        d_Piv[i] = d_Aug[i%Nsize+k*Nsize]/d_Aug[k*(Nsize+1)];
        __syncthreads();
        if (((i%Nsize)>k) && ((i/Nsize/*+1*/)>=k) && ((i/Nsize/*+1*/)<=Nsize))
            d_Aug[i] -= d_Piv[i]*d_Aug[i-(i%Nsize)+k];
        __syncthreads();
    }
}
