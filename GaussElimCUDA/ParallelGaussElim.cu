#include "hip/hip_runtime.h"

/*
 * Device code
 */

__global__ 
void GaussSolve(
         int const Nsize,
         double* d_Aug,
         double* d_Piv)
{
    // Assign matrix elements to blocks and threads
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    // Parallel forward elimination
    for (int k = 0; k < Nsize-1; k++)
    {
        d_Piv[i] = d_Aug[i%Nsize+k*Nsize]/d_Aug[k*(Nsize+1)];
        __syncthreads();
        if (((i%Nsize)>k) && ((i/Nsize/*+1*/)>=k) && ((i/Nsize/*+1*/)<=Nsize))
            d_Aug[i] -= d_Piv[i]*d_Aug[i-(i%Nsize)+k];
        __syncthreads();
    }
}

__global__ void ParallelGaussElim()
{
	// Assign image pixels to blocks and threads
	int i_image = blockDim.y*blockIdx.y + threadIdx.y;
	int j_image = blockDim.x*blockIdx.x + threadIdx.x;

	int offset = (j_image + i_image*nDim_image)*nDim_mat*nDim_mat;

	// Gauss elimination
	for (int k=0; k<nDim-1; k++)
	{
		for (int i=k+1;	i<nDim; i++)
		{
			pivot = d_A[offset+i+k*nDim_mat]/d_A[offset+k+k*nDim_mat];
			for (int j=k; j<nDim; j++)
			{
				d_A[offset+i+j*nDim_mat] -= pivot*d_A[offset+k+j*nDim_mat];
			}
			d_b[offset+i] -= pivot*d_b[offset+i];
		}
	}

/*	do k=1,ndim-1
		do i=k+1,ndim
			pivot=A(i,k)/A(k,k)
			do j=k,ndim
				A(i,j)=A(i,j)-pivot*A(k,j)
			end do
			b(i)=b(i)-pivot*b(k)
		end do
	end do
*/

	// Backward substitution

	for (int i=nDim-1; i>=0; i--)
	{

		for (int j=nDim-1; j>i+1; j--)
		{

		}

	}
/*	do i=ndim,1,-1
		x(i)=b(i)
		do j=ndim,i+1,-1
			x(i)=x(i)-A(i,j)*x(j)
		end do
		x(i)=x(i)/A(i,i)
	end do
*/
}
