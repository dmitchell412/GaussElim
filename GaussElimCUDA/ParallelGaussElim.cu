
#include <hip/hip_runtime.h>

/*
 * Device code
 */

/*
__global__ 
void GaussSolve(
         int const Nsize,
         double* d_Aug,
         double* d_Piv)
{
    for (int i=0; i<16; i++) d_Aug[i]=i;
     Assign matrix elements to blocks and threads
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    // Parallel forward elimination
    for (int k = 0; k < Nsize-1; k++)
    {
        d_Piv[i] = d_Aug[i%Nsize+k*Nsize]/d_Aug[k*(Nsize+1)];
        __syncthreads();
        if (((i%Nsize)>k) && ((i/Nsize)>=k) && ((i/Nsize)<=Nsize))
            d_Aug[i] -= d_Piv[i]*d_Aug[i-(i%Nsize)+k];
        __syncthreads();
    }

}
*/

__global__ void ParallelGaussElim(
	int const nDim_image,
	int const nDim_matrix,
	double* d_A,
	double* d_b,
	double* d_x)
{
	// Assign image pixels to blocks and threads
	int i_image = blockDim.x*blockIdx.x + threadIdx.x;
	//int i_image = blockDim.y*blockIdx.y + threadIdx.y;

	//int offset = (j_image + i_image*nDim_image)*nDim_matrix*nDim_matrix;
	int offset = i_image*nDim_matrix*nDim_matrix;

	// Gauss elimination
	for (int k=0; k<nDim_matrix-1; k++)
	{
		for (int i=k+1;	i<nDim_matrix; i++)
		{
			double pivot = d_A[offset+i+k*nDim_matrix]/d_A[offset+k+k*nDim_matrix];
			for (int j=k; j<nDim_matrix; j++)
			{
				d_A[offset+i+j*nDim_matrix] -= pivot*d_A[offset+k+j*nDim_matrix];
			}
			d_b[offset+i] -= pivot*d_b[offset+k];
		}
	}

/*	do k=1,ndim-1
		do i=k+1,ndim
			pivot=A(i,k)/A(k,k)
			do j=k,ndim
				A(i,j)=A(i,j)-pivot*A(k,j)
			end do
			b(i)=b(i)-pivot*b(k)
		end do
	end do
*/

	// Backward substitution

	for (int i=nDim_matrix-1; i>=0; i--)
	{
		d_x[offset+i] = d_b[offset+i];

		for (int j=nDim_matrix-1; j>i; j--)
		{
			d_x[offset+i] -= d_A[offset+i+j*nDim_matrix]*d_x[offset+j];
		}
        d_x[offset+i] = d_x[offset+i]/d_A[offset+i+i*nDim_matrix];
	}
/*	do i=ndim,1,-1
		x(i)=b(i)
		do j=ndim,i+1,-1
			x(i)=x(i)-A(i,j)*x(j)
		end do
		x(i)=x(i)/A(i,i)
	end do
*/
}
